#include "hip/hip_runtime.h"
//Required for printf()
#include <stdio.h>
//Required for pow(), sqrt()
#include <math.h>

//Represents a point on a
//Euclidean Grid
typedef struct {
	int x;
	int y;
	char zone;

} Point;

// Thread block size
#define BLOCK_SIZE 16

//Prototype for the createVoronoi function.
__global__ void createVoronoi(Point *l_points, int gridWidth, int gridHeight,
		char *l_result, int numPoints);

/*
 * Copies the result array from the GPU after the zone points are calculated.
 * Copies points to the GPU. The kernel finds the Zones in parallel.
 */
void getVoronoiArray(char *result, int gridHeight, int gridWidth, Point *points,
		int numPoints) {

	//Create pointer to char array to hold Zone results
	//Allocate pointer in GPU shared memory
	char *l_result;
	size_t size = (gridWidth * gridHeight) * pow(BLOCK_SIZE, 2) * sizeof(char);
	hipError_t err = hipMalloc(&l_result, size);
	printf("CUDA malloc result array: %s\n", hipGetErrorString(err));

	//Create Point pointer to pass points to GPU shared memory
	Point *l_points;
	err = hipMalloc((void**) &l_points, sizeof(Point) * numPoints);
	printf("CUDA malloc Points: %s\n", hipGetErrorString(err));
	err = hipMemcpy(l_points, points, sizeof(Point) * numPoints,
			hipMemcpyHostToDevice);
	printf("Copy Points to GPU: %s\n", hipGetErrorString(err));


	// Invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(gridWidth, gridHeight);
	createVoronoi<<<dimGrid, dimBlock>>>(l_points, gridWidth, gridHeight, l_result, numPoints);
	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// Read the diagram from GPU into host memory
	err = hipMemcpy(result, l_result, size, hipMemcpyDeviceToHost);
	printf("Copy result from device: %s\n", hipGetErrorString(err));

	// Free device memory
	hipFree(l_result);
	hipFree(l_points);
}

/*
 * Finds the Zone for each thread run. The coordinate generated is based
 * on the (x,y) of the Block, and (x,y) for each thread.
 */
__device__ char getZone(Point *l_points, int x, int y, int numPoints) {
	//Find the first point
	double smallest = sqrt(
			pow((double) l_points[0].x - x, 2)
					+ pow((double) l_points[0].y - y, 2));
	char zone = l_points[0].zone;
	double dist_temp = 0;

	//For each point
	for (int i = 1; i < numPoints; i++) {

		//Find distance to current point
		dist_temp = sqrt(
				pow((double) l_points[i].x - x, 2)
						+ pow((double) l_points[i].y - y, 2));

		//If Point distance is closer,
		//Change the Zone value.
		if (dist_temp < smallest) {
			smallest = dist_temp;
			zone = l_points[i].zone;
		}
	}

	return zone;
}

/*
 *Determines the coordinate of each point in the plane.
 *Sets the result array equal to the appropriate Zone id.
 *Runs in parallel.
 */
__global__ void createVoronoi(Point *l_points, int gridWidth, int gridHeight,
		char *l_result, int numPoints) {

	// X,Y Coordinate of the Block in the defined grid
	int blockCol = blockIdx.x;
	int blockrow = blockIdx.y;

	//X,Y Coordinate of threads in each block
	int row = threadIdx.y;
	int col = threadIdx.x;

	//Find the (x,y) point of the current value
	int x = (blockCol * BLOCK_SIZE) + col;
	int y = (blockrow * BLOCK_SIZE) + row;

	__syncthreads();

	//Set the result array to the proper zone
	l_result[(y * (BLOCK_SIZE * gridWidth)) + x] = getZone(l_points, x, y, numPoints);

}

/*
 * The main method of the program.
 * The program takes the following parameters:
 *
 * int-height int-width int-x1 int-y1 char-y2 xn...
 *
 * Height and width define the result array
 * properties, and (x1,y1) define a Euclidean
 * point, and z1 defines a Zone, which in this
 * case is a single char.
 *
 */
int main(int argc, char* argv[]) {

	//If less than 6, Not enough params to run
	if (argc < 6) {
		printf(
			"Voronoi height, width, x1,y1,z1,x2,y2,z2 ...\nWhere height, width, x, and y are ints\nand z is a single char.");
		return 1;
	}

	//If point params mod 3 does not equal 1
	//There is an unfinished point
	if ((argc - 3) % 3 != 0) {
		printf(
			"Voronoi height, width, x1,y1,z1,x2,y2,z2 ...\nWhere height, width, x, and y are ints\nand z is a single char.");
		return 1;
	}

	//Read height/width of result
	int height = atoi(argv[1]);
	int width = atoi(argv[2]);

	//The total number of points
	int numPoints = (argc - 3) / 3;

	//Create memory allocation for points
	Point * points;
	points = (Point*) malloc(numPoints * sizeof(Point));

	//Read in the point values
	int start = 3;
	for (int i = 0; i < numPoints; i++) {
		points[i].x = atoi(argv[start++]);
		points[i].y = atoi(argv[start++]);
		points[i].zone = argv[start++][0];
	}

	//Grid width - how long the cuda grid must be to obtain result
	int gridWidth = (width / BLOCK_SIZE) + 1;
	//Grid Width - how high the cuda grid must be to obtain result
	int gridHeight = (height / BLOCK_SIZE) + 1;

	//Allocate memory to hold result ( char array )
	char *result;
	result = (char*) malloc(
			(gridWidth * gridHeight) * pow(BLOCK_SIZE, 2) * sizeof(char));

	//Writes the array of zones to the result array
	getVoronoiArray(result, gridHeight, gridWidth, points, numPoints);

	//Shows success!
	printf("Success\n\n");

	//Some information for the user
	printf("Height:%d, Width:%d\n\n", height, width);

	//Prints the values. The lower left is the origin at (0,0).
	int print_width = (BLOCK_SIZE * gridWidth) - width;

	for (int i = height; i > 0; i--) {

		for (int j = gridWidth * BLOCK_SIZE; j > print_width; j--)
			printf("%c ", result[i * (gridWidth * BLOCK_SIZE) - j]);
		printf("\n");
	}

	//Success
	return 1;
}
